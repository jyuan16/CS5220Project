#include "hip/hip_runtime.h"
// Serial implementation so make works
#include "common.h"
#include <cmath>

// Simulation Data Structure
struct person_t
{
    double arrival_time;
    double current_time;
    int queue_id;
    int queue_line;

    person_t(double arrival, double current, int id, int line)
        : arrival_time(arrival), current_time(current), queue_id(id), queue_line(line) {}
};

struct CompareCurrentTime
{
    bool operator()(const person_t &p1, const person_t &p2)
    {
        return p1.current_time > p2.current_time;
    }
};

struct queue_t
{
    int processing_heads;
    int processing_count;
    std::priority_queue<person_t, std::vector<person_t>, CompareCurrentTime> processing_queue;
    std::queue<person_t> waiting_queue;
};

struct airport_t
{
    std::vector<queue_t *> check_in;          // queue_id = 1
    std::vector<queue_t *> bag_check;         // queue_id = 2
    std::vector<queue_t *> security;          // queue_id = 3
    std::vector<queue_t *> security_precheck; // queue_id = 4
} airport;

person_t next_person(0, 0, 0, 0);
std::mt19937 gen(std::random_device{}());
std::uniform_int_distribution<> queue_generator(0, 9);
std::uniform_real_distribution<double> uniform_dist(0.0, 1.0);
std::exponential_distribution<double> entry_dist(entry_rate);
std::normal_distribution<double> check_in_dist(check_in_time, 10);
std::normal_distribution<double> bag_check_dist(bag_check_time, 10);
std::normal_distribution<double> security_dist(security_time, 10);
std::normal_distribution<double> precheck_dist(precheck_time, 3);

void init_simulation()
{
    // Resize the vectors to the number of queues needed for each type of queue
    airport.check_in.resize(num_queue);
    airport.bag_check.resize(num_queue);
    airport.security.resize(num_queue);
    airport.security_precheck.resize(num_queue);

    // Initialize the queues in each vector
    for (int i = 0; i < num_queue; i++)
    {
        airport.check_in[i] = new queue_t;
        airport.check_in[i]->processing_heads = num_check_in;
        airport.check_in[i]->processing_count = 0;

        airport.bag_check[i] = new queue_t;
        airport.bag_check[i]->processing_heads = num_bag_check;
        airport.bag_check[i]->processing_count = 0;

        airport.security[i] = new queue_t;
        airport.security[i]->processing_heads = num_security;
        airport.security[i]->processing_count = 0;

        airport.security_precheck[i] = new queue_t;
        airport.security_precheck[i]->processing_heads = num_precheck;
        airport.security_precheck[i]->processing_count = 0;
    }

    // Schedule the first person to arrive
    double time = entry_dist(gen);
    next_person = person_t(time, time, 0, 0);
}

void add_person(person_t p, queue_t *q, std::normal_distribution<double> dist)
{
    if (q->processing_count < q->processing_heads)
    {
        q->processing_count += 1;
        p.current_time += std::max(0.0, dist(gen));
        q->processing_queue.push(p);
    }
    else
    {
        q->waiting_queue.push(p);
    }
}

void remove_and_update(queue_t *q, std::normal_distribution<double> dist)
{
    q->processing_queue.pop();
    q->processing_count -= 1;
    if (!q->waiting_queue.empty())
    {
        person_t temp = q->waiting_queue.front();
        q->waiting_queue.pop();
        temp.current_time += std::max(0.0, dist(gen));
        q->processing_queue.push(temp);
        q->processing_count += 1;
    }
}

void security_handler(person_t p)
{
    double p_queue = uniform_dist(gen);
    int next_queue = queue_generator(gen);
    if (p_queue < prob_precheck)
    {
        p.queue_id = 4;
        p.queue_line = next_queue;
        add_person(p, airport.security_precheck[next_queue], precheck_dist);
    }
    else
    {
        p.queue_id = 3;
        p.queue_line = next_queue;
        add_person(p, airport.security[next_queue], security_dist);
    }
}

void step(person_t p)
{
    switch (p.queue_id)
    {
    case 0:
    {
        double p_queue = uniform_dist(gen);
        if (p_queue < prob_check_in)
        {
            p.queue_id = 1;
            p.queue_line = queue_generator(gen);
            add_person(p, airport.check_in[p.queue_line], check_in_dist);
        }
        else if (p_queue < prob_check_bag + prob_check_in)
        {
            p.queue_id = 2;
            p.queue_line = queue_generator(gen);
            add_person(p, airport.bag_check[p.queue_line], bag_check_dist);
        }
        else
        {
            security_handler(p);
        }
        break;
    }
    case 1:
    {
        remove_and_update(airport.check_in[p.queue_line], check_in_dist);
        security_handler(p);
        break;
    }
    case 2:
    {
        remove_and_update(airport.bag_check[p.queue_line], bag_check_dist);
        security_handler(p);
        break;
    }
    case 3:
    {
        // std::cout << p.current_time - p.arrival_time << std::endl;
        remove_and_update(airport.security[p.queue_line], security_dist);
        break;
    }
    case 4:
    {
        // std::cout << p.current_time - p.arrival_time << std::endl;
        remove_and_update(airport.security_precheck[p.queue_line], precheck_dist);
        break;
    }
    }
}

void simulate_one_step(double *time)
{
    double next_person_time = next_person.current_time + entry_dist(gen);
    step(next_person);
    next_person = person_t(next_person_time, next_person_time, 0, 0);

    for (auto queue : {airport.check_in, airport.bag_check, airport.security, airport.security_precheck})
    {
        for (int n = 0; n < num_queue; ++n)
        {
            queue_t *q = queue[n];
            if (q->processing_count == q->processing_heads)
            {
                person_t p = q->processing_queue.top();
                step(p);
            }
        }
    }
    for (auto queue : {airport.check_in, airport.bag_check, airport.security, airport.security_precheck})
    {
        for (int n = 0; n < num_queue; ++n)
        {
            if (queue[n]->processing_count > 0)
            {
                if (queue[n]->processing_queue.top().current_time > *time)
                {
                    *time = queue[n]->processing_queue.top().current_time;
                }
            }
        }
    }
}

void run_monte_carlo(int sim_count, int end)
{
    for (int i = 0; i < sim_count; ++i)
    {
        double current_time_val = 0;
        double *current_time = &current_time_val;
        init_simulation();

        while (*current_time < end)
        {
            simulate_one_step(current_time);
        }
    }
}

// #include "common.h"
// #include <cmath>
// #include <hip/hip_runtime.h>
// #include <hiprand/hiprand_kernel.h>

// // Simulation Data Structure
// struct person_t
// {
//     double arrival_time;
//     double current_time;
//     int queue_id;
//     int queue_line;

//     __device__ person_t(double arrival = 0, double current = 0, int id = 0, int line = 0)
//         : arrival_time(arrival), current_time(current), queue_id(id), queue_line(line) {}
// };

// struct queue_t
// {
//     int processing_heads;
//     int processing_count;
//     person_t *processing_queue;
//     int processing_queue_size;
//     person_t *waiting_queue;
//     int waiting_queue_size;
//     int waiting_queue_capacity;

//     __device__ queue_t() : processing_heads(0), processing_count(0), processing_queue(nullptr), processing_queue_size(0),
//                            waiting_queue(nullptr), waiting_queue_size(0), waiting_queue_capacity(0) {}
// };

// struct airport_t
// {
//     queue_t *check_in;
//     queue_t *bag_check;
//     queue_t *security;
//     queue_t *security_precheck;
// } airport;

// __device__ person_t next_person;

// __global__ void setup_kernel(hiprandState *state)
// {
//     int idx = threadIdx.x + blockIdx.x * blockDim.x;
//     hiprand_init(1234, idx, 0, &state[idx]);
// }

// __global__ void init_simulation(queue_t *check_in, queue_t *bag_check, queue_t *security, queue_t *security_precheck)
// {
//     int idx = threadIdx.x + blockIdx.x * blockDim.x;

//     // Allocate memory for queues on the device
//     for (int i = 0; i < num_queue; ++i)
//     {
//         check_in[i].processing_heads = num_check_in;
//         check_in[i].processing_count = 0;
//         check_in[i].processing_queue = new person_t[1000];
//         check_in[i].processing_queue_size = 0;
//         check_in[i].waiting_queue = new person_t[1000];
//         check_in[i].waiting_queue_size = 0;
//         check_in[i].waiting_queue_capacity = 1000;

//         bag_check[i].processing_heads = num_bag_check;
//         bag_check[i].processing_count = 0;
//         bag_check[i].processing_queue = new person_t[1000];
//         bag_check[i].processing_queue_size = 0;
//         bag_check[i].waiting_queue = new person_t[1000];
//         bag_check[i].waiting_queue_size = 0;
//         bag_check[i].waiting_queue_capacity = 1000;

//         security[i].processing_heads = num_security;
//         security[i].processing_count = 0;
//         security[i].processing_queue = new person_t[1000];
//         security[i].processing_queue_size = 0;
//         security[i].waiting_queue = new person_t[1000];
//         security[i].waiting_queue_size = 0;
//         security[i].waiting_queue_capacity = 1000;

//         security_precheck[i].processing_heads = num_precheck;
//         security_precheck[i].processing_count = 0;
//         security_precheck[i].processing_queue = new person_t[1000];
//         security_precheck[i].processing_queue_size = 0;
//         security_precheck[i].waiting_queue = new person_t[1000];
//         security_precheck[i].waiting_queue_size = 0;
//         security_precheck[i].waiting_queue_capacity = 1000;
//     }

//     // Schedule the first person to arrive
//     hiprandState state;
//     hiprand_init(1234, idx, 0, &state);
//     double time = -log(1.0 - hiprand_uniform(&state)) / entry_rate; // Using exponential distribution
//     next_person = person_t(time, time, 0, 0);
// }

// __device__ void add_person(person_t p, queue_t *q, double dist_mean, double dist_stddev, hiprandState *state)
// {
//     double processing_time = hiprand_normal(state) * dist_stddev + dist_mean;
//     if (q->processing_count < q->processing_heads)
//     {
//         q->processing_count += 1;
//         p.current_time += max(0.0, processing_time);
//         q->processing_queue[q->processing_queue_size++] = p;
//     }
//     else
//     {
//         q->waiting_queue[q->waiting_queue_size++] = p;
//     }
// }

// __device__ void remove_and_update(queue_t *q, double dist_mean, double dist_stddev, hiprandState *state)
// {
//     q->processing_queue_size -= 1;
//     q->processing_count -= 1;
//     if (q->waiting_queue_size > 0)
//     {
//         person_t temp = q->waiting_queue[0];
//         for (int i = 1; i < q->waiting_queue_size; ++i)
//         {
//             q->waiting_queue[i - 1] = q->waiting_queue[i];
//         }
//         q->waiting_queue_size -= 1;
//         double processing_time = hiprand_normal(state) * dist_stddev + dist_mean;
//         temp.current_time += max(0.0, processing_time);
//         q->processing_queue[q->processing_queue_size++] = temp;
//         q->processing_count += 1;
//     }
// }

// __device__ void security_handler(person_t p, hiprandState *state)
// {
//     double p_queue = hiprand_uniform(state);
//     int next_queue = hiprand(state) % num_queue;
//     if (p_queue < prob_precheck)
//     {
//         p.queue_id = 4;
//         p.queue_line = next_queue;
//         add_person(p, &airport.security_precheck[next_queue], precheck_time, 3.0, state);
//     }
//     else
//     {
//         p.queue_id = 3;
//         p.queue_line = next_queue;
//         add_person(p, &airport.security[next_queue], security_time, 10.0, state);
//     }
// }

// __device__ void step(person_t p, hiprandState *state)
// {
//     switch (p.queue_id)
//     {
//     case 0:
//     {
//         double p_queue = hiprand_uniform(state);
//         if (p_queue < prob_check_in)
//         {
//             p.queue_id = 1;
//             p.queue_line = hiprand(state) % num_queue;
//             add_person(p, &airport.check_in[p.queue_line], check_in_time, 10.0, state);
//         }
//         else if (p_queue < prob_check_bag + prob_check_in)
//         {
//             p.queue_id = 2;
//             p.queue_line = hiprand(state) % num_queue;
//             add_person(p, &airport.bag_check[p.queue_line], bag_check_time, 10.0, state);
//         }
//         else
//         {
//             security_handler(p, state);
//         }
//         break;
//     }
//     case 1:
//     {
//         remove_and_update(&airport.check_in[p.queue_line], check_in_time, 10.0, state);
//         security_handler(p, state);
//         break;
//     }
//     case 2:
//     {
//         remove_and_update(&airport.bag_check[p.queue_line], bag_check_time, 10.0, state);
//         security_handler(p, state);
//         break;
//     }
//     case 3:
//     {
//         remove_and_update(&airport.security[p.queue_line], security_time, 10.0, state);
//         break;
//     }
//     case 4:
//     {
//         remove_and_update(&airport.security_precheck[p.queue_line], precheck_time, 3.0, state);
//         break;
//     }
//     }
// }

// __global__ void simulate_one_step(double *time, hiprandState *states)
// {
//     int idx = threadIdx.x + blockIdx.x * blockDim.x;
//     hiprandState state = states[idx];
//     double next_person_time = next_person.current_time + -log(1.0 - hiprand_uniform(&state)) / entry_rate;
//     step(next_person, &state);
//     next_person = person_t(next_person_time, next_person_time, 0, 0);

//     for (int n = 0; n < num_queue; ++n)
//     {
//         if (airport.check_in[n].processing_count == airport.check_in[n].processing_heads)
//         {
//             person_t p = airport.check_in[n].processing_queue[0];
//             step(p, &state);
//         }
//         if (airport.bag_check[n].processing_count == airport.bag_check[n].processing_heads)
//         {
//             person_t p = airport.bag_check[n].processing_queue[0];
//             step(p, &state);
//         }
//         if (airport.security[n].processing_count == airport.security[n].processing_heads)
//         {
//             person_t p = airport.security[n].processing_queue[0];
//             step(p, &state);
//         }
//         if (airport.security_precheck[n].processing_count == airport.security_precheck[n].processing_heads)
//         {
//             person_t p = airport.security_precheck[n].processing_queue[0];
//             step(p, &state);
//         }
//     }

//     for (int n = 0; n < num_queue; ++n)
//     {
//         if (airport.check_in[n].processing_count > 0 && airport.check_in[n].processing_queue[0].current_time > *time)
//         {
//             *time = airport.check_in[n].processing_queue[0].current_time;
//         }
//         if (airport.bag_check[n].processing_count > 0 && airport.bag_check[n].processing_queue[0].current_time > *time)
//         {
//             *time = airport.bag_check[n].processing_queue[0].current_time;
//         }
//         if (airport.security[n].processing_count > 0 && airport.security[n].processing_queue[0].current_time > *time)
//         {
//             *time = airport.security[n].processing_queue[0].current_time;
//         }
//         if (airport.security_precheck[n].processing_count > 0 && airport.security_precheck[n].processing_queue[0].current_time > *time)
//         {
//             *time = airport.security_precheck[n].processing_queue[0].current_time;
//         }
//     }
//     states[idx] = state;
// }

// void run_monte_carlo(int sim_count, int end)
// {
//     // Allocate memory for queues on the host
//     queue_t *h_check_in = (queue_t *)malloc(num_queue * sizeof(queue_t));
//     queue_t *h_bag_check = (queue_t *)malloc(num_queue * sizeof(queue_t));
//     queue_t *h_security = (queue_t *)malloc(num_queue * sizeof(queue_t));
//     queue_t *h_security_precheck = (queue_t *)malloc(num_queue * sizeof(queue_t));

//     // Allocate memory for queues on the device
//     queue_t *d_check_in;
//     queue_t *d_bag_check;
//     queue_t *d_security;
//     queue_t *d_security_precheck;
//     hipMalloc((void **)&d_check_in, num_queue * sizeof(queue_t));
//     hipMalloc((void **)&d_bag_check, num_queue * sizeof(queue_t));
//     hipMalloc((void **)&d_security, num_queue * sizeof(queue_t));
//     hipMalloc((void **)&d_security_precheck, num_queue * sizeof(queue_t));

//     // Initialize the simulation
//     init_simulation<<<1, 1>>>(d_check_in, d_bag_check, d_security, d_security_precheck);
//     hipMemcpy(h_check_in, d_check_in, num_queue * sizeof(queue_t), hipMemcpyDeviceToHost);
//     hipMemcpy(h_bag_check, d_bag_check, num_queue * sizeof(queue_t), hipMemcpyDeviceToHost);
//     hipMemcpy(h_security, d_security, num_queue * sizeof(queue_t), hipMemcpyDeviceToHost);
//     hipMemcpy(h_security_precheck, d_security_precheck, num_queue * sizeof(queue_t), hipMemcpyDeviceToHost);

//     // Setup hiprand states
//     hiprandState *d_states;
//     hipMalloc((void **)&d_states, sim_count * sizeof(hiprandState));
//     setup_kernel<<<1, sim_count>>>(d_states);

//     for (int i = 0; i < sim_count; ++i)
//     {
//         double current_time_val = 0;
//         double *d_current_time;
//         hipMalloc((void **)&d_current_time, sizeof(double));
//         hipMemcpy(d_current_time, &current_time_val, sizeof(double), hipMemcpyHostToDevice);

//         while (current_time_val < end)
//         {
//             simulate_one_step<<<1, sim_count>>>(d_current_time, d_states);
//             hipMemcpy(&current_time_val, d_current_time, sizeof(double), hipMemcpyDeviceToHost);
//         }

//         hipFree(d_current_time);
//     }
// }
