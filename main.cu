#include "common.h"
#include <chrono>
#include <cmath>
#include <cstring>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <random>
#include <vector>

// Command Line Option Processing
int find_arg_idx(int argc, char **argv, const char *option)
{
    for (int i = 1; i < argc; ++i)
    {
        if (strcmp(argv[i], option) == 0)
        {
            return i;
        }
    }
    return -1;
}

int find_int_arg(int argc, char **argv, const char *option, int default_value)
{
    int iplace = find_arg_idx(argc, argv, option);

    if (iplace >= 0 && iplace < argc - 1)
    {
        return std::stoi(argv[iplace + 1]);
    }

    return default_value;
}

char *find_string_option(int argc, char **argv, const char *option, char *default_value)
{
    int iplace = find_arg_idx(argc, argv, option);

    if (iplace >= 0 && iplace < argc - 1)
    {
        return argv[iplace + 1];
    }

    return default_value;
}

// ==============
// Main Function
// ==============

int main(int argc, char **argv)
{
    // Parse Args
    if (find_arg_idx(argc, argv, "-h") >= 0)
    {
        std::cout << "Options:" << std::endl;
        std::cout << "-h: see this help" << std::endl;
        std::cout << "-d: <int>: set end time (in seconds)" << std::endl;
        return 0;
    }

    // Initialize Particles
    int end = find_int_arg(argc, argv, "-d", 1) * (24 * 60 * 60);

    // Algorithm
    auto start_time = std::chrono::steady_clock::now();

    init_simulation();
    double current_time_val = 0;
    double *current_time = &current_time_val;

    while (*current_time < end)
    {
        simulate_one_step(current_time);
        hipDeviceSynchronize();
    }

    hipDeviceSynchronize();
    auto end_time = std::chrono::steady_clock::now();

    std::chrono::duration<double> diff = end_time - start_time;
    double seconds = diff.count();

    // Finalize
    std::cout << "Simulation Time = " << seconds << " seconds for " << end / (24 * 60 * 60) << " days.\n";
}
